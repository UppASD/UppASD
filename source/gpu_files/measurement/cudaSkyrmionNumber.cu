#include "hip/hip_runtime.h"
#include "cudaSkyrmionNumber.cuh"
#include "fortranData.hpp"
#include "cudaParallelizationHelper.hpp"


__global__ void grad_moments_kernel(const CudaTensor<real, 3> emomM,
                                    const CudaTensor<real, 3> dxyz_vec,
                                    const CudaTensor<int, 2> dxyz_atom,
                                    const CudaTensor<int, 1> dxyz_list,
                                    CudaTensor<real, 4> grad_mom);

__global__ void pontryagin_no_kernel(const CudaTensor<real, 3> emomM,
                                     const CudaTensor<real, 4> grad_mom,
                                     real* pontryagin_no_out);

__global__ void avrg_skyno_kernel();


SkyrmionNumber::SkyrmionNumber(const CudaTensor<real, 3>& emomM, const CudaTensor<real, 3>& emom)
: emomM(emomM)
, emom(emom)
, skyno_step(*FortranData::skyno_step)
, buffer_size(*FortranData::skyno_buff)
{
    const uint N = emomM.extent(1);
    const uint M = emomM.extent(2);

    Tensor<real, 3> dxyz_vec_fortran(FortranData::dxyz_vec, 3, 26, N);
    dxyz_vec.Allocate(3, 26, N); // why 26?
    dxyz_vec.copy_sync(dxyz_vec_fortran);

    Tensor<int, 2> dxyz_atom_fortran(FortranData::dxyz_atom, 26, N);
    dxyz_atom.Allocate(26, N);
    dxyz_atom.copy_sync(dxyz_atom_fortran);

    Tensor<int, 1> dxyz_list_fortran(FortranData::dxyz_list, N);
    dxyz_list.Allocate(N);
    dxyz_list.copy_sync(dxyz_list_fortran);

//    std::cout << "dxyz_vec\n";
//    for (uint i = 0; i < N; ++i)
//    {
//        std::cout << i << ": ";
//        for (uint k = 0; k < 26; ++k)
//        {
//            std::cout << "(" << dxyz_vec_fortran(0, k, i) << " "
//                      << dxyz_vec_fortran(1, k, i) << " "
//                      << dxyz_vec_fortran(2, k, i) << "), ";
//        }
//        std::cout << "\n";
//    }
//
//    std::cout << "\ndxyz_atom\n";
//    for (uint i = 0; i < N; ++i)
//    {
//        std::cout << i << ": ";
//        for (uint k = 0; k < 26; ++k)
//        {
//            std::cout << dxyz_atom_fortran(k, i) << " ";
//        }
//        std::cout << "\n";
//    }
//
//    std::cout << "\ndxyz_list\n";
//    for (uint i = 0; i < N; ++i)
//    {
//        std::cout << i << ": " << dxyz_list_fortran(i) << "\n";
//    }

    grad_mom.Allocate(3, 3, N, M);
    grad_mom.zeros();

    skynob.Allocate(buffer_size);
    skynob.zeros();

    sk_avrg.Allocate(buffer_size);
    sk_avrg.zeros();

    sk_var.Allocate(buffer_size);
    sk_var.zeros();

    // TODO: this should not be allocated, but hooked up to fortran buffer pointer
    indxb_skyno.AllocateHost(buffer_size);
}


SkyrmionNumber::~SkyrmionNumber()
{
    dxyz_vec.Free();
    dxyz_atom.Free();
    dxyz_list.Free();
    grad_mom.Free();
    skynob.Free();
    sk_avrg.Free();
    sk_var.Free();
    indxb_skyno.FreeHost();
}


void SkyrmionNumber::measure(std::size_t mstep)
{
    --mstep;

    if (mstep % skyno_step != 0)
        return;

    // std::cout << "[SkyrmionNumber::measure] mstep = " << mstep << ", ";

    const uint N = emomM.extent(1);
    const uint M = emomM.extent(2);

    // this seems to be done on every time step in fortran
    hipStream_t workStream = CudaParallelizationHelper::def.getWorkStream();
    dim3 threads = {512};
    dim3 blocks = {
            (N + threads.x - 1) / threads.x,
            (M + threads.y - 1) / threads.y
    };

    // TODO are we sure this should be emom, and not emomM?
    grad_moments_kernel<<<blocks, threads, 0, workStream>>>(emom, dxyz_vec, dxyz_atom, dxyz_list, grad_mom);
    pontryagin_no_kernel<<<blocks, threads, 0, workStream>>>(emomM, grad_mom, skynob.data() + buffer_count);
    indxb_skyno(buffer_count++) = static_cast<uint>(mstep);

    // hipDeviceSynchronize();

    if (buffer_count >= buffer_size)
    {
        // TODO: copy to fortran
        buffer_count = 0;
        skynob.zeros();
    }
}


void SkyrmionNumber::flushMeasurements(std::size_t mstep)
{

}


__global__ void grad_moments_kernel(const CudaTensor<real, 3> emomM,
                                    const CudaTensor<real, 3> dxyz_vec,
                                    const CudaTensor<int, 2> dxyz_atom,
                                    const CudaTensor<int, 1> dxyz_list,
                                    CudaTensor<real, 4> grad_mom)
{
    const uint N = emomM.extent(1);
    const uint M = emomM.extent(2);
    const uint iatom = blockDim.x * blockIdx.x + threadIdx.x;
    const uint kk = blockDim.y * blockIdx.y + threadIdx.y;

    if (iatom >= N || kk >= M)
        return;

    assert(dxyz_list(iatom) < N);

    for (uint jneigh = 0; jneigh < dxyz_list(iatom); ++jneigh)
    {
        assert(jneigh < 26);
        const uint jatom = dxyz_atom(jneigh, iatom) - 1; // needs -1 here since it gives the index of a neighboring atom

        assert(jatom < N);
        const real d_mom[3] = {
            emomM(0, jatom, kk) - emomM(0, iatom, kk),
            emomM(1, jatom, kk) - emomM(1, iatom, kk),
            emomM(2, jatom, kk) - emomM(2, iatom, kk)
        };

        const real dv[3] = { // dv = {dx, dy, dz}
            dxyz_vec(0, jneigh, iatom),
            dxyz_vec(1, jneigh, iatom),
            dxyz_vec(2, jneigh, iatom)
        };

        for (uint coord = 0; coord < 3; ++coord)
        {
            if (abs( dv[coord] ) > 1e-7)
            {
                grad_mom(0, coord, iatom, kk) += d_mom[0] / dv[coord];
                grad_mom(1, coord, iatom, kk) += d_mom[1] / dv[coord];
                grad_mom(2, coord, iatom, kk) += d_mom[2] / dv[coord];
            }
        }
    }

    for (uint coord = 0; coord < 3; ++coord)
    {
        grad_mom(coord, 0, iatom, kk) /= dxyz_list(iatom);
        grad_mom(coord, 1, iatom, kk) /= dxyz_list(iatom);
        grad_mom(coord, 2, iatom, kk) /= dxyz_list(iatom);
    }
}


__global__ void pontryagin_no_kernel(const CudaTensor<real, 3> emomM,
                                     const CudaTensor<real, 4> grad_mom,
                                     real* pontryagin_no_out)
{
    const uint N = emomM.extent(1);
    const uint M = emomM.extent(2);

    const uint iatom = blockDim.x * blockIdx.x + threadIdx.x;
    const uint k = blockDim.y * blockIdx.y + threadIdx.y;

    if (iatom >= N || k >= M)
        return;

    const real cvec_x = grad_mom(1,0,iatom,k) * grad_mom(2,1,iatom,k)
                      - grad_mom(2,0,iatom,k) * grad_mom(1,1,iatom,k);

    const real cvec_y = grad_mom(2,0,iatom,k) * grad_mom(0,1,iatom,k)
                      - grad_mom(0,0,iatom,k) * grad_mom(2,1,iatom,k);

    const real cvec_z = grad_mom(0,0,iatom,k) * grad_mom(1,1,iatom,k)
                      - grad_mom(1,0,iatom,k) * grad_mom(0,1,iatom,k);


    const real partial_sum = emomM(0,iatom,k) * cvec_x
                           + emomM(1,iatom,k) * cvec_y
                           + emomM(2,iatom,k) * cvec_z;

    atomicAdd(pontryagin_no_out, partial_sum);

    __syncthreads();

    if (iatom == 0 && k == 0)
    {
        *pontryagin_no_out /= (M_PI * M);
    }
}


__global__ void avrg_skyno_kernel(CudaTensor<real, 1> skynob,
                                  CudaTensor<real, 1> sk_avrg,
                                  CudaTensor<real, 1> sk_var)
{
    const uint buffer_count = skynob.extent(0);
    for (uint k = 0; k < buffer_count; ++k)
    {

    }
}














