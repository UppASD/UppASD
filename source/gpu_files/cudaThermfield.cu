#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "c_headers.hpp"
#include "cudaMatrix.hpp"
#include "cudaParallelizationHelper.hpp"
#include "cudaThermfield.hpp"
#include "fortMatrix.hpp"
#include "real_type.h"
#include "stopwatch.hpp"
#include "stopwatchDeviceSync.hpp"
#include "stopwatchPool.hpp"

////////////////////////////////////////////////////////////////////////////////
// Parallelization helper classes
////////////////////////////////////////////////////////////////////////////////


// The neighbour list setup helper
class CudaThermfield::SetupSigmaFactor : public CudaParallelizationHelper::Site {
private:
   real* sigma_factor;
   real dp;

public:
   SetupSigmaFactor(real* p1, real p2) {
      sigma_factor = p1;
      dp = p2;
   }

   __device__ void each(unsigned int site) {
      sigma_factor[site] = sqrt(dp * sigma_factor[site]);
   }
};


class CudaThermfield::SetupField : public CudaParallelizationHelper::AtomSite {
private:
   real* field;
   const real* sigma_factor;
   const real* mmom;

public:
   SetupField(real* p1, const real* p2, const real* p3) {
      field = p1;
      sigma_factor = p2;
      mmom = p3;
   }

   __device__ void each(unsigned int atom, unsigned int site) {
      real sigma = sigma_factor[site] * rsqrt(mmom[atom]);
      field[atom * 3 + 0] *= sigma;
      field[atom * 3 + 1] *= sigma;
      field[atom * 3 + 2] *= sigma;
   }
};


////////////////////////////////////////////////////////////////////////////////
// Class members
////////////////////////////////////////////////////////////////////////////////
CudaThermfield::CudaThermfield()
    : stopwatch(GlobalStopwatchPool::get("Cuda thermfield")),
      parallel(CudaParallelizationHelper::def) {
   constantsInitiated = false;
   dataInitiated = false;
}


CudaThermfield::~CudaThermfield() {
   if(dataInitiated) {
      hiprandDestroyGenerator(gen);
   }
}


bool CudaThermfield::initiate(std::size_t N, std::size_t M, hiprandRngType_t rngType,
                              unsigned long long seed) {
   if(dataInitiated) {
      std::fprintf(stderr, "Warning: attempt to initiate already initiated CudaThermfield\n");
      return true;
   }

   stopwatch.skip();

   if(field.initiate(3, N, M) && sigmaFactor.initiate(N)) {
      if(hiprandCreateGenerator(&gen, rngType) == HIPRAND_STATUS_SUCCESS) {
         if(seed == 0ULL) {
            seed = time(nullptr);
         }
         hiprandSetPseudoRandomGeneratorSeed(gen, seed);
         hiprandSetStream(gen, parallel.getWorkStream());
         dataInitiated = true;
      } else {
         field.free();
         sigmaFactor.free();
      }
   }
   stopwatch.add("initiate");
   return dataInitiated;
}


bool CudaThermfield::initiateConstants(const fortMatrix<real, 1>& temperature, real timestep, real gamma,
                                       real k_bolt, real mub, real damping) {
   // Timing
   stopwatch.skip();

   // Initiated?
   if(!dataInitiated) {
      return false;
   }

   // Damping parameter
   real dp = (2.0 * damping * k_bolt) / (timestep * gamma * mub * (1 + damping * damping));

   // Set up sigmaFactor
   sigmaFactor.memcopy(temperature, parallel.getWorkStream());

   // sF = sqrt(dp*sF) ( = sqrt(dp*temp))
   parallel.cudaSiteCall(SetupSigmaFactor(sigmaFactor, dp));
   stopwatch.add("initiate constants");

   constantsInitiated = true;
   return true;
}


void CudaThermfield::randomize(const cudaMatrix<real, 2>& mmom) {
   // Initiated?
   if(!initiated()) {
      return;
   }

   // Timing
   stopwatch.skip();

// Generate random vector
#ifdef SINGLE_PREC
   hiprandGenerateNormal(gen, field.get_data(), field.size(), 0.0, 1.0);
#else
   hiprandGenerateNormalDouble(gen, field.get_data(), field.size(), 0.0, 1.0);
#endif
   stopwatch.add("RNG");

   // Expand thermal field
   parallel.cudaAtomSiteCall(SetupField(field, sigmaFactor, mmom));
   stopwatch.add("loop");
}

