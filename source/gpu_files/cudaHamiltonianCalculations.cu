#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>

using namespace std;

#include "real_type.h"
#include <iostream>

#include "hostMatrix.hpp"
#include "cudaMatrix.hpp"

#include "cudaHamiltonianCalculations.hpp"

// Possible improvements

////////////////////////////////////////////////////////////////////////////////
// Parallelization helper classes
////////////////////////////////////////////////////////////////////////////////

// The neighbour list setup helper
//
// Note (Thomas):
// For Heisenberg 
// Class sets everything between neighbours and maxneighbours
// to zero since hamiltonian implementation always runs to max neighbours
class CudaHamiltonianCalculations::SetupNeighbourList :
	public CudaParallelizationHelper::Site {
private:
	real *               coup;
	unsigned int *       pos;
	const unsigned int * size;
	unsigned int         mnn;

public:
	SetupNeighbourList(const Exchange &ex) {
		coup  = ex.coupling;
		size  = ex.neighbourCount;
		pos   = ex.neighbourPos;
		mnn   = ex.mnn;
	}

        __device__ void each(unsigned int site) {
		real *         myCoup = &coup[site];
		unsigned int * myPos  = &pos[site];
		unsigned int   mySize = size[site];
		for (unsigned int i = 0; i < mnn; i++) {

			if (i < mySize)
				myPos[i * N]--;
			else {
				myCoup[i * N] = (real)0.0;
				myPos[i * N]  = 0;
			}
		}
	}
};


// The neighbour list setup helper
//
// For Tensorial Exchange 
class CudaHamiltonianCalculations::SetupNeighbourListExchangeTensor :
	public CudaParallelizationHelper::Site {
private:
	real *               tensor;
	unsigned int *       pos;
	const unsigned int * size;
	unsigned int         mnn;

public:
	SetupNeighbourListExchangeTensor(const TensorialExchange &tenEx) {
		
		tensor  = tenEx.tensor;
		size  = tenEx.neighbourCount;
		pos   = tenEx.neighbourPos;
		mnn   = tenEx.mnn;

	}

        __device__ void each(unsigned int site) {
		//real *         myCoup = &coup[site];
		unsigned int * myPos  = &pos[site];
		unsigned int   y = size[site];

		for (unsigned int i = 0; i < mnn; i++) { 

			if (pos[site * mnn + i] != 0)
			{
				pos[site * mnn + i]--;
			}
			else {
				pos[site * mnn + i] = 0;

				unsigned int k = i;
				unsigned int l = site;

				// Dimension of the tensorial exchange matrix: (dim1,dim2,dim3,dim4)  <--> (3,3,mnn,N)
				// Calculating the matrix elements of the exchange tensor and setting them to zero:
			    tensor[0 + 3 * (0 + 3 * (k + mnn * l))] = (real)0.0; // i=0,j=0
			    tensor[0 + 3 * (1 + 3 * (k + mnn * l))] = (real)0.0; // i=0,j=1		
			    tensor[0 + 3 * (2 + 3 * (k + mnn * l))] = (real)0.0; // i=0,j=2		
			    tensor[1 + 3 * (0 + 3 * (k + mnn * l))] = (real)0.0; // i=1,j=0		
			    tensor[1 + 3 * (1 + 3 * (k + mnn * l))] = (real)0.0; // i=1,j=1		
			    tensor[1 + 3 * (2 + 3 * (k + mnn * l))] = (real)0.0; // i=1,j=2		
			    tensor[2 + 3 * (0 + 3 * (k + mnn * l))] = (real)0.0; // i=2,j=0		
			    tensor[2 + 3 * (1 + 3 * (k + mnn * l))] = (real)0.0; // i=2,j=1		
			    tensor[2 + 3 * (2 + 3 * (k + mnn * l))] = (real)0.0; // i=2,j=2	
			}
		}
	}
};



// unnecessary for anisotropy probably
class CudaHamiltonianCalculations::SetupAnisotropy :
	public CudaParallelizationHelper::Site {
private:
	real * kaniso;
	real * eaniso;
	unsigned int * taniso;
public:
	SetupAnisotropy(const Anisotropy &aniso) {

		kaniso = aniso.kaniso;
		eaniso = aniso.eaniso;
		taniso = aniso.taniso;
	}
        __device__ void each(unsigned int site) {
		
	}
};


// Note (Thomas):
// For DM interaction
// Class sets everything between neighbours and maxneighbours
// to zero since hamiltonian implementation always runs to max neighbours
class CudaHamiltonianCalculations::SetupNeighbourListDM :
	public CudaParallelizationHelper::Site {
private:
	real *               coup;
	unsigned int *       pos;
	const unsigned int * size;
	unsigned int         mnn;

public:
	SetupNeighbourListDM(const DMinteraction & dm) {
		coup  = dm.interaction;
		size  = dm.neighbourCount;
		pos   = dm.neighbourPos;
		mnn   = dm.mnn;
	}

        __device__ void each(unsigned int site) {

		// Phil's 
		for (unsigned int i = 0; i < mnn; i++) { 

			if (pos[site * mnn + i] != 0)
			{
				pos[site * mnn + i]--;
			}
			else {
				pos[site * mnn + i] = 0;

				unsigned int k = i;
				unsigned int l = site;

				// Dimension of the DM vector: (dim1,dim2,dim3)  <--> (3,mnn,N)
				coup[0 + 3 * i + site * mnn * 3] = (real)0.0;
				coup[1 + 3 * i + site * mnn * 3] = (real)0.0;
				coup[2 + 3 * i + site * mnn * 3] = (real)0.0;
			}
		}



		// DM code which is not workung properly
		//real *         myCoup = &coup[site * 3];
		//unsigned int * myPos  = &pos[site];
		//unsigned int   mySize = size[site];
		//for (unsigned int i = 0; i < mnn; i++) {
		//	if (i < mySize)
		//		myPos[i * N]--;
		//	else {
		//		myCoup[i * N + 0] = (real)0.0;
		//		myCoup[i * N + 1] = (real)0.0;
		//		myCoup[i * N + 2] = (real)0.0;
		//		myPos[i * N]      = 0;
		//	}
		//}
	}
				

};

// Note: (Thomas)
// Calculating the magnetic field from various effects
// such as the heisenberg field and DM interactions
// Added DM effect 2014/09/23
class CudaHamiltonianCalculations::HeisgeJij :
	public CudaParallelizationHelper::AtomSiteEnsemble {
private:
	real *               beff;
	const real *         coup;
	const unsigned int * pos;
	const real *         emomM;
	const real *         ext_f;
	unsigned int         mnn;
	const real *         dmcoup;
	const unsigned int * dmpos;
	unsigned int         dmmnn;
public:
	HeisgeJij(real * p1, const real * p2, const real * p3,
			 const Exchange & ex, const DMinteraction & dm) {
		beff   = p1;
		emomM  = p2;
		ext_f  = p3;

		coup   = ex.coupling;
		pos    = ex.neighbourPos;
		mnn    = ex.mnn;

		dmcoup = dm.interaction;
		dmpos  = dm.neighbourPos;
		dmmnn   = dm.mnn; 
	}

        __device__ void each(unsigned int atom, unsigned int site, unsigned int ensemble) {
		// Field
		real x = (real)0.0;
		real y = (real)0.0;
		real z = (real)0.0;

		// Pointers with fixed indices
		const real *         site_coup = &coup[site];
		const unsigned int * site_pos  = &pos[site];
		const real *         my_emomM  = &emomM[ensemble * N * 3];

		const real *         site_dmcoup    = &dmcoup[site];
		const unsigned int * site_dmpos     = &dmpos[site];

		// Exchange term loop
		for (unsigned int i = 0; i < mnn; i++) {
			unsigned int x_offset = site_pos[i * N] * 3; 
			real c = site_coup[i * N];
			x += c * my_emomM[x_offset + 0];
			y += c * my_emomM[x_offset + 1];
			z += c * my_emomM[x_offset + 2];
		}

		// Phil's DM interaction implementation (still only incorporated into the isotropic Heisenberg exchange)
		for (unsigned int i = 0; i < dmmnn; i++){

			unsigned int neighborPosIndex = dmpos[site * dmmnn + i]; // neighbor position in the site enemble given in 0,1,2,...,N-1

			unsigned int x_offset = neighborPosIndex * 3; 

			real Sx = my_emomM[x_offset + 0];
			real Sy = my_emomM[x_offset + 1];
			real Sz = my_emomM[x_offset + 2];
			real Dx = dmcoup[0 + 3 * i + site * dmmnn * 3];
			real Dy = dmcoup[1 + 3 * i + site * dmmnn * 3];
			real Dz = dmcoup[2 + 3 * i + site * dmmnn * 3];

			x += - Dz * Sy + Dy * Sz;
			y += - Dx * Sz + Dz * Sx;
			z += - Dy * Sx + Dx * Sy;
		}


		// DM interaction, almost no performance impact if dmmnn is 0	
		//for (unsigned int i = 0; i < dmmnn; i++) {
		//	unsigned int x_offset = site_dmpos[i * N] * 3; 
		//	x += -site_dmcoup[i*N+2]*my_emomM[x_offset+1] + site_dmcoup[i*N+1]*my_emomM[x_offset+2];
		//	y += -site_dmcoup[i*N+0]*my_emomM[x_offset+2] + site_dmcoup[i*N+2]*my_emomM[x_offset+0];
		//	z += -site_dmcoup[i*N+1]*my_emomM[x_offset+0] + site_dmcoup[i*N+0]*my_emomM[x_offset+1];
		//}

		// Save field
		beff[atom * 3 + 0] = x + ext_f[atom * 3 + 0];
		beff[atom * 3 + 1] = y + ext_f[atom * 3 + 1];
		beff[atom * 3 + 2] = z + ext_f[atom * 3 + 2];
	}
};


class CudaHamiltonianCalculations::HeisJijTensor :
	public CudaParallelizationHelper::AtomSiteEnsemble {
private:
	real *               beff;
	const real *         tensor;
	const unsigned int * pos;
	const unsigned int * size;
	const real *         emomM;
	const real *         ext_f;
	unsigned int         mnn;
public:
	HeisJijTensor(real * p1, const real * p2, const real * p3,
			 const TensorialExchange &tenEx) {
		beff   = p1;
		emomM  = p2;
		ext_f  = p3;
		
		tensor = tenEx.tensor;
		pos    = tenEx.neighbourPos;
		size   = tenEx.neighbourCount;
		mnn    = tenEx.mnn;
	}

        __device__ void each(unsigned int atom, unsigned int site, unsigned int ensemble) {
		// Field
		real x = (real)0.0;
		real y = (real)0.0;
		real z = (real)0.0;

		// Pointers with fixed indices
		const unsigned int * site_pos  = &pos[site];
		const real *         my_emomM  = &emomM[ensemble * N * 3];


		// emomM <--> (3,N,M)
		// tensor <---> (3,3,mnn,N)
		// pos   <--> (mnn,N)

		// Tensorial exchange coupling
		for (unsigned int i = 0; i < mnn; i++) {


			unsigned int neighborPosIndex = pos[site * mnn + i]; // neighbor position in the site enemble given in 0,1,2,...,N-1

			unsigned int x_offset = neighborPosIndex * 3; 

			unsigned int k = i;
			unsigned int l = site;

			real J11 = tensor[0 + 3 * (0 + 3 * (k + mnn * l))]; // i=0,j=0
			real J12 = tensor[0 + 3 * (1 + 3 * (k + mnn * l))]; // i=0,j=1		
			real J13 = tensor[0 + 3 * (2 + 3 * (k + mnn * l))]; // i=0,j=2		
			real J21 = tensor[1 + 3 * (0 + 3 * (k + mnn * l))]; // i=1,j=0		
			real J22 = tensor[1 + 3 * (1 + 3 * (k + mnn * l))]; // i=1,j=1		
			real J23 = tensor[1 + 3 * (2 + 3 * (k + mnn * l))]; // i=1,j=2		
			real J31 = tensor[2 + 3 * (0 + 3 * (k + mnn * l))]; // i=2,j=0		
			real J32 = tensor[2 + 3 * (1 + 3 * (k + mnn * l))]; // i=2,j=1		
			real J33 = tensor[2 + 3 * (2 + 3 * (k + mnn * l))]; // i=2,j=2	

			// magnetic moment of current neighbor
			real Sx = my_emomM[x_offset + 0];
			real Sy = my_emomM[x_offset + 1];
			real Sz = my_emomM[x_offset + 2];

			x += J11 * Sx + J12 * Sy + J13 * Sz;
			y += J21 * Sx + J22 * Sy + J23 * Sz;
			z += J31 * Sx + J32 * Sy + J33 * Sz;
		}

		// Save field
		beff[atom * 3 + 0] = x + ext_f[atom * 3 + 0];
		beff[atom * 3 + 1] = y + ext_f[atom * 3 + 1];
		beff[atom * 3 + 2] = z + ext_f[atom * 3 + 2];
	}
};


class CudaHamiltonianCalculations::HeisgeJijAniso :
	public CudaParallelizationHelper::AtomSiteEnsemble {
private:
	real *               beff;
	const real *         emomM;
	const real * kaniso;
	const real * eaniso;
	const unsigned int * taniso;
	const real * sb;
public:
	HeisgeJijAniso(real * p1, const real * p2,
			 const Anisotropy & aniso) {
		beff   = p1;
		emomM  = p2;
		kaniso = aniso.kaniso;
		eaniso = aniso.eaniso;
		taniso = aniso.taniso;
		sb     = aniso.sb;
	}

        __device__ void each(unsigned int atom, unsigned int site, unsigned int ensemble) {
		// Field
		real x = (real)0.0;
		real y = (real)0.0;
		real z = (real)0.0;
		// Magnetic moment at current site/atom
		real Sx = (real)0.0;
		real Sy = (real)0.0;
		real Sz = (real)0.0;
		// Uniaxial anisotropy unit vector
		real ex = (real)0.0;
		real ey = (real)0.0;
		real ez = (real)0.0;
		const real *  my_emomM  = &emomM[ensemble * N * 3];

		const unsigned int type = taniso[site]; // type of the anisotropy: 0 = none, 1 = uniaxial, 2 = cubic



		Sx = emomM[atom * 3 + 0];
		Sy = emomM[atom * 3 + 1];
		Sz = emomM[atom * 3 + 2];

		// direction of uniaxial anisotropy
		ex = eaniso[0 + site * 3];
		ey = eaniso[1 + site * 3];
		ez = eaniso[2 + site * 3];

		// anisotropy constants
		const real k1 = kaniso[0 + site * 2];
		const real k2 = kaniso[1 + site * 2]; 
		
		if(type == 1 || type == 7)  // uniaxial anisotropy
		{
			const real tt1 = Sx * ex + Sy * ey + Sz * ez;
			const real tt2 = k1 + (real)2.0 * k2 * (1-tt1*tt1);
			const real tt3 = (real)2.0*tt1*tt2;

			x += -tt3*ex;
			y += -tt3*ey;
			z += -tt3*ez;
		}   
		if (type == 2 || type == 7) { // cubic anisotropy

			real k1_cubic = k1;
			real k2_cubic = k2;

			if (type == 7) { // Apply uniaxial and cubic anisotropy: The Cubic Anisotropy constant = Uniaxial constant x sb
				k1_cubic *= sb[site];
				k2_cubic *= sb[site];
			}

			x += (real)2.0*k1_cubic*Sx*(Sy*Sy+Sz*Sz) + (real)2.0*k2_cubic*Sx*Sy*Sy*Sz*Sz;
			y += (real)2.0*k1_cubic*Sy*(Sz*Sz+Sx*Sx) + (real)2.0*k2_cubic*Sy*Sz*Sz*Sx*Sx;
			z += (real)2.0*k1_cubic*Sz*(Sx*Sx+Sy*Sy) + (real)2.0*k2_cubic*Sz*Sx*Sx*Sy*Sy;
		}

		// Save field
		beff[atom * 3 + 0] += x;
		beff[atom * 3 + 1] += y;
		beff[atom * 3 + 2] += z;
	}
};



class CudaHamiltonianCalculations::HeisgeJijElement :
	public CudaParallelizationHelper::ElementAxisSiteEnsemble {
private:
	real *               beff;
	const real *         coup;
	const unsigned int * pos;
	const unsigned int * size;
	const real *         emomM;
	const real *         ext_f;
	unsigned int         mnn;
public:
	HeisgeJijElement(real * p1, const real * p5, const real * p6, const Exchange & ex) {
		beff   = p1;
		coup   = ex.coupling;
		pos    = ex.neighbourPos;
		size   = ex.neighbourCount;
		emomM  = p5;
		ext_f  = p6;
		mnn    = ex.mnn;
	}

        __device__ void each(unsigned int element, unsigned int axis, unsigned int site, unsigned int ensemble) {
		// Field
		real f = (real)0.0;

		// Pointers with fixed indices
		const real *         site_coup      = &coup[site];
		const unsigned int * site_pos       = &pos[site];
		const real *         ensemble_emomM = &emomM[ensemble * N * 3];

		// Exchange term loop
//		const unsigned int s = size[i];
//		for (int j = 0; j < s; j++) {
		for (unsigned int i = 0; i < mnn; i++) {
			unsigned int offset = site_pos[i * N] * 3;
			f += site_coup[i * N] * ensemble_emomM[offset + axis];
		}

		// Save field
		beff[element] = f + ext_f[element];
	}
};


////////////////////////////////////////////////////////////////////////////////
// Helpers
////////////////////////////////////////////////////////////////////////////////
template<typename T>
static void transpose(T * A, const T * B, size_t M, size_t N) {
	for (size_t y = 0; y < M; ++y)
		for (size_t x = 0; x < N; ++x)
			A[(x * M) + y] = B[(y * N) + x];
}

template <typename T, size_t I, size_t J, size_t K>
static void transpose(hostMatrix<T,2,I,J,K> &A, const hostMatrix<T,2,I,J,K> &B) {
	// Sizes
	size_t M = A.dimension_size(0);
	size_t N = A.dimension_size(1);

	if (B.dimension_size(1) != M || B.dimension_size(0) != N) {
		fprintf(stderr, "Error: illegal matrix transpose\n");
		exit(EXIT_FAILURE);
	}

	transpose(A.get_data(), B.get_data(), M, N);
}

// Function for testing time impact of optimal neighbour alignment
// Will not produce correct results
void alignOptimal(hostMatrix<unsigned int,2> &nlist, bool same) {
	// Sizes
	size_t N   = nlist.dimension_size(0);
	size_t mnn = nlist.dimension_size(1);

	for (size_t m = 0; m < mnn; ++m)
		for (size_t n = 0; n < N; ++n)
			nlist(n,m) = same ? ((m % N) + 1) : (((n + 32 * m) % N) + 1);
}





////////////////////////////////////////////////////////////////////////////////
// Class members
////////////////////////////////////////////////////////////////////////////////

CudaHamiltonianCalculations::CudaHamiltonianCalculations() :
	parallel(CudaParallelizationHelper::def) {
	initiated = false;
}

bool CudaHamiltonianCalculations::initiate(
		const hostMatrix<real,2>         &ncoup, 
		const hostMatrix<unsigned int,2> &nlist,
		const hostMatrix<unsigned int,1> &nlistsize,
		const hostMatrix<real,3,3>       &dm_ncoup, 
		const hostMatrix<unsigned int,2> &dm_nlist,
		const hostMatrix<unsigned int,1> &dm_nlistsize,
		const int 			 do_dm,
		const int do_j_tensor,
		const hostMatrix<real,4,3,3> j_tensor,
		const int do_aniso,
		const hostMatrix<real,2,2> kaniso,
		const hostMatrix<real,2,3> eaniso,
		const hostMatrix<unsigned int, 1> taniso, 
		const hostMatrix<real, 1> sb) {

	// Memory access is better if N is multiple of 32
	// (alignment of 128 bytes, see Cuda Best Parctice Guide)
	N      = ncoup.dimension_size(1);    // Number of atoms
	if (N % 32 != 0) {
		printf("Note: Performance is better if the number of atoms is a multiple of 32.\n");
	}


	//------- Anisotropy -------//
	if (do_aniso != 0) {
		aniso.kaniso.clone(kaniso);
		aniso.eaniso.clone(eaniso);
		aniso.taniso.clone(taniso);
		aniso.sb.clone(sb);
		CudaHamiltonianCalculations::do_aniso = do_aniso;
	}


	//------- Tensorial Exchange -------//
	if (do_j_tensor == 1)
	{
		CudaHamiltonianCalculations::do_j_tensor = true;
		
		N = j_tensor.dimension_size(3);

		// Matrixes are not transposed when using tensorial exchange
		//hostMatrix<real,4,3,3>         j_tensor_t;
		//hostMatrix<unsigned int,2> nlist_t;
		//j_tensor_t.initiate(3,3,N,tenEx.mnn);
		//nlist_t.initiate(N,tenEx.mnn);
		//transpose(j_tensor_t, j_tensor);
		//transpose(nlist_t, nlist);

		tenEx.mnn = j_tensor.dimension_size(2);
		tenEx.neighbourCount.clone(nlistsize);
		tenEx.neighbourPos.clone(nlist);
		tenEx.tensor.clone(j_tensor);

		//for(unsigned int site = 0; site < N; site++) {
		//	const unsigned int * myPos  = &(nlist.get_data())[site];
		//	const unsigned int   mySize = nlistsize.get_data()[site];
		//	printf(" %d ", myPos[0]);
		//	printf("| ");
		//	for (unsigned int i = 0; i < tenEx.mnn; i++)
		//	{
		//		printf(" %d ", myPos[i * N]);
		//	}
		//	printf("\n");
		//}

		parallel.cudaSiteCall(SetupNeighbourListExchangeTensor(tenEx));

		// Did we get the memory?
		if (!tenEx.tensor.has_data()       ||
			!tenEx.neighbourCount.has_data() ||
			!tenEx.neighbourPos.has_data()) {
			release();
			return false;
		}
		// Flag
		initiated = true;
		return true;
	}

	//------- Heisenberg Exchange -------//
	ex.mnn    = ncoup.dimension_size(0);    // Max number of neighbours

	// Transposing the matrices will make CUDA calculations faster
	hostMatrix<real,2>         ncoup_t;
	hostMatrix<unsigned int,2> nlist_t;

	ncoup_t.initiate(N,ex.mnn);
	nlist_t.initiate(N,ex.mnn);

	transpose(ncoup_t, ncoup);
	transpose(nlist_t, nlist);

	// TEST
	//alignOptimal(nlist_t, true);
	//printf("blubb: %f",ex.coupling);

	ex.coupling.clone(ncoup_t);
	ex.neighbourCount.clone(nlistsize);
	ex.neighbourPos.clone(nlist_t);

	// Did we get the memory?
	if (!ex.coupling.has_data()       ||
	    !ex.neighbourCount.has_data() ||
	    !ex.neighbourPos.has_data()) {
		release();
		return false;
	}

	// List setup kernel call
	parallel.cudaSiteCall(SetupNeighbourList(ex));

	//------- DM Interaction -------//
	dm.mnn = 0;
	if (do_dm) {
		dm.mnn = dm_ncoup.dimension_size(1); // Max number of DM neighbours  // I CHANGED THE INDEX FROM 0 TO 1!!!

		dm.interaction.clone(dm_ncoup);
		dm.neighbourCount.clone(dm_nlistsize);
		dm.neighbourPos.clone(dm_nlist);
		
		if (!dm.interaction.has_data()       ||
			!dm.neighbourCount.has_data() ||
			!dm.neighbourPos.has_data()) {
			release();
			return false;
		}
		parallel.cudaSiteCall(SetupNeighbourListDM(dm));
	}

	// Flag
	initiated = true;
	return true;
}


void CudaHamiltonianCalculations::release() {
	ex.coupling.free();
	ex.neighbourCount.free();
	ex.neighbourPos.free();
	dm.interaction.free();
	dm.neighbourCount.free();
	dm.neighbourPos.free();
	initiated = false;
}

void CudaHamiltonianCalculations::heisge(cudaMatrix<real,3,3> &beff, 
		const cudaMatrix<real,3,3> &emomM,
		const cudaMatrix<real,3,3> &external_field) {
	
	// Kernel call

	if (do_j_tensor == 1)
	{
		parallel.cudaAtomSiteEnsembleCall(HeisJijTensor(beff, emomM, external_field, tenEx));

	} else {
		parallel.cudaAtomSiteEnsembleCall(HeisgeJij(beff, emomM, external_field, ex, dm));
	}

	if (do_aniso != 0)
	{
		parallel.cudaAtomSiteEnsembleCall(HeisgeJijAniso(beff, emomM, aniso));
	}



	return;




	//parallel.cudaElementAxisSiteEnsembleCall(HeisgeJijElement(beff, emomM, external_field, ex));
}
